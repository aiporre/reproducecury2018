#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "CudaKernels.h"

///////////////////////////////////////
///// Conv1D ////////////////////////////
///////////////////////////////////////


// thread kernel: computation of gammai = sum_j k(xi,yj)betaj for index i given by thread id.


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER  >
__global__ void GpuConv1DOnDevice(KER Ker,
                                      TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
                                      int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], gammai[DIMVECT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<ny) // we load yj and betaj from device global memory only if j<ny
        {
            int inc = DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        
        if(i<nx) // we compute gammai only if needed
        {
            TYPE *yj, *betaj;
            yj = SharedData;
            betaj = SharedData + DIMPOINT;
            int inc = DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
		Ker.Eval(gammai,xi,yj,betaj);
    	}
	__syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMVECT; k++)
            gamma[i*DIMVECT+k] = gammai[k];
}



template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuEvalConv1D(KER Ker, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
    // Data on the device.
    TYPE* x_d;
    TYPE* y_d;
    TYPE* beta_d;
    TYPE* gamma_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

    // Compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
	
	GpuConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMVECT+DIMPOINT)*sizeof(TYPE)>>>
			(Ker, x_d, y_d, beta_d, gamma_d, nx, ny);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}



template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuEvalConv1D(TYPE sigma, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	
	return GpuEvalConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)), 
			x_h, y_h, beta_h, gamma_h, nx, ny);
}



template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuEvalConv1D(TYPE sigma, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	
	return GpuEvalConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			x_h, y_h, beta_h, gamma_h, nx, ny);
}



////////////////////////////////////////
///// GRAD1 Conv1D ///////////////////////
////////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGrad1Conv1DOnDevice(KER Ker,
        TYPE *alpha, TYPE *x, TYPE *y, TYPE *beta, TYPE *gamma,
        int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], alphai[DIMVECT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi and alphai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            alphai[k] = alpha[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<ny) // we load yj and betaj from device global memory only if j<ny
        {
            int inc = DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = y[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *yj, *betaj;
            yj = SharedData;
            betaj = SharedData + DIMPOINT;
            int inc = DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc)
	            Ker.Grad1(gammai,alphai,xi,yj,betaj);
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gamma[i*DIMPOINT+k] = gammai[k];
}

//////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGrad1Conv1D(KER Ker, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* y_d;
    TYPE* alpha_d;
    TYPE* gamma_d;
    TYPE* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMPOINT));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(ny*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, sizeof(TYPE)*(ny*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(ny*DIMVECT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    GpuGrad1Conv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
		<<<gridSize,blockSize,blockSize.x*(DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
			(Ker, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(alpha_d);
    hipFree(gamma_d);
    hipFree(beta_d);

    return 0;
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuGrad1Conv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	return GpuGrad1Conv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)), 
			alpha_h, x_h, y_h, beta_h, gamma_h, nx, ny);
}


template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGrad1Conv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int nx, int ny)
{
	return GpuGrad1Conv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			alpha_h, x_h, y_h, beta_h, gamma_h, nx, ny);
}


///////////////////////////////////////
////////// GRAD Conv1D //////////////////
///////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradConv1DOnDevice(KER Ker,
        TYPE *alpha, TYPE *x, TYPE *beta, TYPE *gamma,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], alphai[DIMVECT], betai[DIMVECT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, alphai, betai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            alphai[k] = alpha[i*DIMVECT+k];
        for(int k=0; k<DIMVECT; k++)
            betai[k] = beta[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, alphaj and betaj from device global memory only if j<nx
        {
            int inc = DIMPOINT + 2 * DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = alpha[j*DIMVECT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *alphaj, *betaj;
            xj = SharedData;
            alphaj = SharedData + DIMPOINT;
            betaj = SharedData + DIMPOINT + DIMVECT;
            int inc = DIMPOINT + 2 * DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, alphaj+=inc, betaj+=inc)
                Ker.Grad(gammai, xi, xj, alphai, alphaj, betai, betaj);
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gamma[i*DIMPOINT+k] = gammai[k];
}

////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradConv1D(KER Ker,
        TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h,
         int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* alpha_d;
    TYPE* gamma_d;
    TYPE* beta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&alpha_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(alpha_d, alpha_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	GpuGradConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, alpha_d, x_d, beta_d, gamma_d, nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(alpha_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuGradConv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h, int nx)
{
	return GpuGradConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)), 
			alpha_h, x_h, beta_h, gamma_h, nx);
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradConv1D(TYPE sigma, TYPE* alpha_h, TYPE* x_h, TYPE* beta_h, TYPE* gamma_h, int nx)
{
	return GpuGradConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			alpha_h, x_h, beta_h, gamma_h, nx);
}

////////////////////////////////////////////
////////// GRAD DIFF Conv1D //////////////////
////////////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuGradDiffConv1DOnDevice(KER Ker,
        TYPE *x, TYPE *beta, TYPE *eta, TYPE *gamma,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], betai[DIMVECT], etai[DIMPOINT], gammai[DIMPOINT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, etai, betai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            betai[k] = beta[i*DIMVECT+k];
        for(int k=0; k<DIMPOINT; k++)
            etai[k] = eta[i*DIMPOINT+k];
        for(int k=0; k<DIMPOINT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, etaj and betaj from device global memory only if j<nx
        {
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = eta[j*DIMPOINT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *betaj, *etaj;
            xj = SharedData;
            betaj = SharedData + DIMPOINT;
            etaj = SharedData + DIMPOINT + DIMVECT;
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, betaj+=inc, etaj+=inc)
                Ker.GradDiff(gammai, xi, xj, betai, betaj, etai, etaj);
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMPOINT; k++)
            gamma[i*DIMPOINT+k] = gammai[k];
}

////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuGradDiffConv1D(KER Ker,
        TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h,
         int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* beta_d;
    TYPE* gamma_d;
    TYPE* eta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&eta_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMPOINT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(eta_d, eta_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    GpuGradDiffConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, x_d, beta_d, eta_d, gamma_d, nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMPOINT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(eta_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuGradDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuGradDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)), 
			x_h, beta_h, eta_h, gamma_h, nx);
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuGradDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuGradDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)),
			x_h, beta_h, eta_h, gamma_h, nx);
}


////////////////////////////////////////////
////////// DIFF Conv1D ///////////////////////
////////////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
__global__ void GpuDiffConv1DOnDevice(KER Ker,
        TYPE *x, TYPE *beta, TYPE *eta, TYPE *gamma,
        int nx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);

    TYPE xi[DIMPOINT], etai[DIMPOINT], gammai[DIMVECT];
    if(i<nx)  // we will compute gammai only if i is in the range
    {
        // load xi, etai from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k] = x[i*DIMPOINT+k];
        for(int k=0; k<DIMPOINT; k++)
            etai[k] = eta[i*DIMPOINT+k];
        for(int k=0; k<DIMVECT; k++)
            gammai[k] = 0.0f;
    }

    for(int jstart = 0, tile = 0; jstart < nx; jstart += blockDim.x, tile++)
    {
        int j = tile * blockDim.x + threadIdx.x;
        if(j<nx) // we load xj, betaj and etaj from device global memory only if j<nx
        {
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k] = x[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] = eta[j*DIMPOINT+k];
        }
        __syncthreads();
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *xj, *betaj, *etaj;
            xj = SharedData;
            betaj = SharedData + DIMPOINT;
            etaj = SharedData + DIMPOINT + DIMVECT;
            int inc = 2 * DIMPOINT + DIMVECT;
            for(int jrel = 0; jrel < blockDim.x && jrel<nx-jstart; jrel++, xj+=inc, betaj+=inc, etaj+=inc)
                Ker.Diff(gammai, xi, xj, betaj, etai, etaj);
        }
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMVECT; k++)
            gamma[i*DIMVECT+k] = gammai[k];
}

////////////////////////////////////////////////////////////////////////////

template < typename TYPE, int DIMPOINT, int DIMVECT, class KER >
int GpuDiffConv1D(KER Ker,
        TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h,
        int nx)
{

    // Data on the device.
    TYPE* x_d;
    TYPE* beta_d;
    TYPE* gamma_d;
    TYPE* eta_d;

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&beta_d, sizeof(TYPE)*(nx*DIMVECT));
    hipMalloc((void**)&eta_d, sizeof(TYPE)*(nx*DIMPOINT));
    hipMalloc((void**)&gamma_d, sizeof(TYPE)*(nx*DIMVECT));

    // Send data from host to device.
    hipMemcpy(x_d, x_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);
    hipMemcpy(beta_d, beta_h, sizeof(TYPE)*(nx*DIMVECT), hipMemcpyHostToDevice);
    hipMemcpy(eta_d, eta_h, sizeof(TYPE)*(nx*DIMPOINT), hipMemcpyHostToDevice);

    // compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

    GpuDiffConv1DOnDevice<TYPE,DIMPOINT,DIMVECT,KER>
        <<<gridSize,blockSize,blockSize.x*(2*DIMPOINT+DIMVECT)*sizeof(TYPE)>>>
            (Ker, x_d, beta_d, eta_d, gamma_d, nx);

    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(gamma_h, gamma_d, sizeof(TYPE)*(nx*DIMVECT),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(eta_d);
    hipFree(beta_d);
    hipFree(gamma_d);

    return 0;
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int CauchyGpuDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,CauchyFunction<TYPE> >(CauchyFunction<TYPE>(sigma)), 
			x_h, beta_h, eta_h, gamma_h, nx);
}

template < typename TYPE, int DIMPOINT, int DIMVECT >
int GaussGpuDiffConv1D(TYPE sigma, TYPE* x_h, TYPE* beta_h, TYPE* eta_h, TYPE* gamma_h, int nx)
{
	return GpuDiffConv1D < TYPE, DIMPOINT, DIMVECT, ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> > >
		(ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,GaussFunction<TYPE> >(GaussFunction<TYPE>(sigma)), 
			x_h, beta_h, eta_h, gamma_h, nx);
}




// http://www.parashift.com/c++-faq-lite/separate-template-fn-defn-from-decl.html
#define DECLARE_Conv1DS(TYPE,DIMPOINT,DIMVECT) \
	template int CauchyGpuEvalConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int CauchyGpuGrad1Conv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int CauchyGpuGradConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int CauchyGpuGradDiffConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int CauchyGpuDiffConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int GaussGpuEvalConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int GaussGpuGrad1Conv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, TYPE*, int, int); \
	template int GaussGpuGradConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int GaussGpuGradDiffConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); \
	template int GaussGpuDiffConv1D<TYPE,DIMPOINT,DIMVECT>(TYPE, TYPE*, TYPE*, TYPE*, TYPE*, int); 
#define DECLARE_Conv1DS_ALLDIMVECTS_FOR(TYPE,DIMPOINT) \
        DECLARE_Conv1DS(TYPE,DIMPOINT,1) \
        DECLARE_Conv1DS(TYPE,DIMPOINT,2) \
        DECLARE_Conv1DS(TYPE,DIMPOINT,3)
#define DECLARE_Conv1DS_ALLDIMS_FOR(TYPE) \
        DECLARE_Conv1DS_ALLDIMVECTS_FOR(TYPE,1) \
        DECLARE_Conv1DS_ALLDIMVECTS_FOR(TYPE,2) \
        DECLARE_Conv1DS_ALLDIMVECTS_FOR(TYPE,3)
DECLARE_Conv1DS_ALLDIMS_FOR(double)
DECLARE_Conv1DS_ALLDIMS_FOR(float)




