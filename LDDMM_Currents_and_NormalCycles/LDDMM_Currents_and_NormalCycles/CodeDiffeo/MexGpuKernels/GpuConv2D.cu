#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "Pack.h"


using namespace std;

template <typename TYPE, int DIMVECT>
__global__ void reduce0(TYPE* in, TYPE* out, int sizeY,int nx)
{
	TYPE res = 0;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < nx*DIMVECT)
    {
		for (int i = 0; i < sizeY; i++) 
            res += in[tid + i*nx*DIMVECT];
		/*res = in[tid+ nx* DIMVECT];*/
		out[tid] = res;
	}
}








// thread kernel: computation of x1i = sum_j k(x2i,x3i,...,y1j,y2j,...) for index i given by thread id.
template < typename TYPE, class KER, class FUN >
__global__ void GpuConv2DOnDevice(KER Ker, FUN fun, int nx, int ny, TYPE** px, TYPE** py)
{

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    const int DIMX = DIMSX::SUM;
    const int DIMY = DIMSY::SUM;        
    const int DIMX1 = DIMSX::FIRST;

    extern __shared__ char yj_char[];
    TYPE* const yj = reinterpret_cast<TYPE*>(yj_char);

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    TYPE xi[DIMX];
    if(i<nx)  // we will compute x1i only if i is in the range
    {
        for(int k=0; k<DIMX1; k++)
            xi[k] = 0.0f;
        // load xi from device global memory
	DIMSX::NEXT::load(i,xi+DIMX1,px+1);
    }
    
    int j = blockIdx.y * blockDim.x + threadIdx.x;
    if(j<ny) // we load yj from device global memory only if j<ny
	DIMSY::load(j,yj+threadIdx.x*DIMY,py);    	
    __syncthreads();
        
    if(i<nx) // we compute x1i only if needed
    {
    	TYPE* yjrel = yj;
        for(int jrel = 0; (jrel<blockDim.x) && ((blockDim.x*blockIdx.y+jrel)< ny); jrel++, yjrel+=DIMY)
		DIMSX::template call<DIMSY>(fun,xi,yjrel,Ker);	
        __syncthreads();
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMX1; k++)
            (*px)[blockIdx.y*DIMX1*nx+i*DIMX1+k] = xi[k];
}
///////////////////////////////////////////////////


template < typename TYPE, class KER, class FUN >
int GpuConv2D(KER Ker, FUN fun, int nx, int ny, TYPE** px_h, TYPE** py_h)
{
    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    const int DIMX = DIMSX::SUM;
    const int DIMY = DIMSY::SUM;
    const int DIMX1 = DIMSX::FIRST;
    const int SIZEX = DIMSX::SIZE;
    const int SIZEY = DIMSY::SIZE;

    // Data on the device.
    TYPE *x1B, *x_d, *y_d, **px_d, **py_d;

    hipHostAlloc((void**)&px_d, SIZEX*sizeof(TYPE*), hipHostMallocMapped);
    hipHostAlloc((void**)&py_d, SIZEY*sizeof(TYPE*), hipHostMallocMapped);

    // Allocate arrays on device.
    hipMalloc((void**)&x_d, sizeof(TYPE)*(nx*DIMX));
    hipMalloc((void**)&y_d, sizeof(TYPE)*(ny*DIMY));

    // Send data from host to device.

    int nvals;
    px_d[0] = x_d;
    nvals = nx*DIMSX::VAL(0);
    for(int k=1; k<SIZEX; k++)
    {
        px_d[k] = px_d[k-1] + nvals;
        nvals = nx*DIMSX::VAL(k);
        hipMemcpy(px_d[k], px_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }
    py_d[0] = y_d;
    nvals = ny*DIMSY::VAL(0);
    hipMemcpy(py_d[0], py_h[0], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    for(int k=1; k<SIZEY; k++)
    {
        py_d[k] = py_d[k-1] + nvals;
        nvals = ny*DIMSY::VAL(k);
        hipMemcpy(py_d[k], py_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice);
    }

    // Compute on device.
    dim3 blockSize;
    blockSize.x = 192; // number of threads in each block
    int blockSizey = blockSize.x;
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
	gridSize.y =  ny / blockSizey + (ny%blockSizey==0 ? 0 : 1);

    // Reduce  : grid and block are 1d
    dim3 blockSize2;
    blockSize2.x = 192; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMX1) / blockSize2.x + ((nx*DIMX1)%blockSize2.x==0 ? 0 : 1);

    hipMalloc((void**)&x1B, sizeof(TYPE)*(nx*DIMX1*gridSize.y));
    px_d[0] = x1B;

    GpuConv2DOnDevice<TYPE><<<gridSize,blockSize,blockSize.x*(DIMY)*sizeof(TYPE)>>>(Ker,fun,nx,ny,px_d,py_d);
	

    reduce0<TYPE,DIMX1><<<gridSize2, blockSize2>>>(x1B, x_d, gridSize.y,nx);
    
    // block until the device has completed
    hipDeviceSynchronize();

    // Send data from device to host.
    hipMemcpy(*px_h, x_d, sizeof(TYPE)*(nx*DIMX1),hipMemcpyDeviceToHost);

    // Free memory.
    hipFree(x_d);
    hipFree(y_d);
    hipFree(x1B);
    hipFree(px_d);
    hipFree(py_d);

    return 0;
}


template < typename TYPE, class KER, class FUN, typename... Args >
int GpuConv2D(KER Ker, FUN fun, int nx, int ny, TYPE* x1_h, Args... args)
{

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    const int SIZEX = DIMSX::SIZE;
    const int SIZEY = DIMSY::SIZE;

    TYPE *px_h[SIZEX];
    TYPE *py_h[SIZEY];
    DIMSX::getlist(px_h,x1_h,args...);
    DIMSX::template getlist<DIMSY>(py_h,x1_h,args...);
 
	return GpuConv2D(Ker,fun,nx,ny,x1_h,px_h,py_h);

}



