#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <mex.h>
#include "GpuConv2D.cu"
#include "CudaKernels.h"
#include "CudaNCSurfKernels.h"
#include "CudaVarSurfKernels.h"

void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{

    typedef RADIALFUN<__TYPE__ > RadialFun;

#define SCALARRADIAL 1
#if KERNEL==SCALARRADIAL
typedef ScalarRadialKernel<__TYPE__,__DIMPOINT__,__DIMVECT__,RadialFun> KER;
#endif

#define NCSURF 2
#if KERNEL==NCSURF
typedef NCSurfKernel<__TYPE__,RadialFun> KER;
#endif

#define VARSURF 3
#if KERNEL==VARSURF
typedef VarSurfKernel<__TYPE__,RadialFun> KER;
#endif

struct KER::EVAL funeval;

    const int DIMX1 = KER::EVAL::DIMSX::FIRST;
    typedef typename KER::EVAL::DIMSX DIMSX;   
    typedef typename KER::EVAL::DIMSY DIMSY;
    const int SIZEX = DIMSX::SIZE;
    const int SIZEY = DIMSY::SIZE;        

    if(nrhs<SIZEX+SIZEY-1)
        mexErrMsgIdAndTxt("KernelGpuConv:nrhs","At least %d inputs required.",SIZEX+SIZEY-1);

    if(nrhs>SIZEX+SIZEY)
        mexErrMsgIdAndTxt("KernelGpuConv:nrhs","Too many inputs.");

    double *x[SIZEX];
    int nx[SIZEX];

    double *y[SIZEY];
    int ny[SIZEY];

    for(int k=1; k<SIZEX; k++)
    {
        x[k] = mxGetPr(prhs[k-1]);
        if(mxGetM(prhs[k-1])!=DIMSX::VAL(k))
            mexErrMsgIdAndTxt("KernelGpuConv:inputdims","Dimensions of inputs are not compatible.");
        nx[k] = mxGetN(prhs[k-1]);
        if(nx[k]!=nx[1])
            mexErrMsgIdAndTxt("KernelGpuConv:inputdims","Dimensions of inputs are not compatible.");
    }

    for(int k=0; k<SIZEY; k++)
    {
        y[k] = mxGetPr(prhs[SIZEX+k-1]);
        if(mxGetM(prhs[SIZEX+k-1])!=DIMSY::VAL(k))
            mexErrMsgIdAndTxt("KernelGpuConv:inputdims","Dimensions of inputs are not compatible.");
        ny[k] = mxGetN(prhs[SIZEX+k-1]);
        if(ny[k]!=ny[0])
            mexErrMsgIdAndTxt("KernelGpuConv:inputdims","Dimensions of inputs are not compatible.");
    }

    nx[0] = nx[1];
    plhs[0] = mxCreateDoubleMatrix((mwSize)DIMX1,(mwSize)nx[0],mxREAL);
    x[0] = mxGetPr(plhs[0]);

    if(nrhs==SIZEX+SIZEY)
    {
        int deviceID = *mxGetPr(prhs[SIZEX+SIZEY-1]);
        hipSetDevice(deviceID);
    }

	GpuConv2D(KER(RadialFun()),funeval,nx[0],ny[0],x,y);

}

